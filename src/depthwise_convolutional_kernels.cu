#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "depthwise_convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

	__global__ void DepthwiseConv2dGPUKernelNCHW(
		const float* input,const int in_rows, const int in_cols, const int in_depth,
		const float* filter, const int filter_rows, const int filter_cols,
		const int stride,const int pad_rows,const int pad_cols,
		const int out_rows,const int out_cols,const int out_depth,
		float* output, int num_outputs) {


	int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (thread_id >= num_outputs) return;


	//计算当前output像素点的四维索引索引
	const int OC = thread_id % out_cols;//width
	const int OR = (thread_id / out_cols) % out_rows;//height
	const int OD = (thread_id / out_cols / out_rows) % out_depth;//channel
	const int OB = thread_id / out_cols / out_rows / out_depth;//batch size

	const int in_d = OD ;



	const int input_offset_temp = (OB * in_depth + in_d) * (in_rows * in_cols);//当前output channel对应的input channel 的指针


	const int input_row_start = OR * stride - pad_rows;
	const int input_col_start = OC * stride - pad_cols;
	const int input_row_end = input_row_start + filter_rows;
	const int input_col_end = input_col_start + filter_cols;

	float sum = 0;
	if (input_row_start >= 0 && input_col_start >= 0 &&
		input_row_end < in_rows && input_col_end < in_cols)
	{
		#pragma unroll
			for (int f_r = 0; f_r < filter_rows; ++f_r) {
				const int in_r = input_row_start + f_r;
				#pragma unroll
				for (int f_c = 0; f_c < filter_cols; ++f_c) {
					const int in_c = input_col_start + f_c;

					const int input_offset =
						(input_offset_temp)+(in_r * in_cols) + in_c;
					const int filter_offset =f_c + filter_cols * f_r +OD*filter_cols*filter_rows;
					sum += (*(input + input_offset)) * (*(filter + filter_offset));
				}
			}
		}
	else {
		#pragma unroll
		for (int f_r = 0; f_r < filter_rows; ++f_r) {
				const int in_r = input_row_start + f_r;
				#pragma unroll
				for (int f_c = 0; f_c < filter_cols; ++f_c) {
					const int in_c = input_col_start + f_c;

					if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
						const int in_c = input_col_start + f_c;

						const int input_offset =
							(input_offset_temp)+(in_r * in_cols) + in_c;

						const int filter_offset = f_c + filter_cols * f_r + OD*filter_cols*filter_rows;
						sum += (*(input + input_offset)) * (*(filter + filter_offset));
					}
				}
			}
		}

	output[thread_id] = sum;

}

__global__ void DepthwiseConv2dBackpropFilterGPUKernelNCHW(const float* out_backprop,
			const int stride, const int pad_rows, const int pad_cols, const int out_rows, const int out_cols, const int out_depth,
			const float* input, const int in_rows, const int in_cols, const int in_depth,
			float* filter_backprop, const int filter_rows, const int filter_cols,
			int num_out_backprop) {

	int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (thread_id >= num_out_backprop) return;


	const int out_c = thread_id % out_cols;
	const int out_r = (thread_id / out_cols) % out_rows;
	const int out_d = (thread_id / out_cols / out_rows) % out_depth;

	const int b = thread_id / out_depth / out_cols / out_rows;
	const int in_d = out_d;


	const int in_r_start = out_r * stride - pad_rows;
	const int in_c_start = out_c * stride - pad_cols;
	const int in_r_end = in_r_start + filter_rows;
	const int in_c_end = in_c_start + filter_cols;

	const int out_backprop_offset = (b * out_depth * out_rows * out_cols) +
				(out_d * out_rows * out_cols) +(out_r * out_cols) + (out_c);

	const float out_bp = *(out_backprop + out_backprop_offset);
	if (in_r_start >= 0 && in_c_start >= 0 && in_r_end < in_rows &&in_c_end < in_cols) {
		#pragma unroll 
		for (int f_r = 0; f_r < filter_rows; ++f_r) {
			const int in_r = in_r_start + f_r;
			const int input_offset_temp = (b * in_depth * in_rows * in_cols) +
						(in_d * in_rows * in_cols) +(in_r * in_cols);

			#pragma unroll 
			for (int f_c = 0; f_c < filter_cols; ++f_c) {
						const int in_c = in_c_start + f_c;
						const int input_offset = input_offset_temp + in_c;
						float partial_sum = (*(input + input_offset)) * out_bp;
						float* addr = filter_backprop + f_c + filter_cols * f_r + out_d*filter_cols*filter_rows;
						atomicAdd(addr, partial_sum);
					}
				}
			}
			else {
				#pragma unroll 
				for (int f_r = 0; f_r < filter_rows; ++f_r) {
					const int in_r = in_r_start + f_r;
					const int input_offset_temp = (b * in_depth * in_rows * in_cols) +(in_d * in_rows * in_cols) +(in_r * in_cols);
					#pragma unroll 
					for (int f_c = 0; f_c < filter_cols; ++f_c) {
						const int in_c = in_c_start + f_c;
						if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols) {
							const int input_offset = input_offset_temp + in_c;
							float partial_sum = (*(input + input_offset)) * out_bp;
							float* addr =filter_backprop + f_c + filter_cols * f_r + out_d*filter_cols*filter_rows;
							atomicAdd(addr, partial_sum);
						}
					}
				}

		}
	}



__global__ void DepthwiseConv2dBackpropInputGPUKernelNCHW(
		const float* out_backprop, const int out_rows, const int out_cols, const int out_depth,
		const float* filter, const int filter_rows, const int filter_cols,
		float* in_backprop, const int in_rows, const int in_cols, const int in_depth,
		const int stride, const int pad_rows, const int pad_cols,int num_in_backprop)
{
		int thread_id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
		if (thread_id >= num_in_backprop) return;

		const int in_c = thread_id % in_cols;
		const int in_r = (thread_id / in_cols) % in_rows;
		const int in_d = (thread_id / in_cols / in_rows) % in_depth;
		const int b = thread_id / in_depth / in_cols / in_rows;

		float sum = 0;


		const int out_r_start =max(0, (in_r - filter_rows + pad_rows + stride) / stride);
		const int out_r_end = min(out_rows - 1, (in_r + pad_rows) / stride);
		const int out_c_start =
			max(0, (in_c - filter_cols + pad_cols + stride) / stride);
		const int out_c_end = min(out_cols - 1, (in_c + pad_cols) / stride);


		#pragma unroll 
			for (int out_r = out_r_start; out_r <= out_r_end; ++out_r) {
				const int f_r = in_r + pad_rows - out_r * stride;


				for (int out_c = out_c_start; out_c <= out_c_end; ++out_c) {
					const int f_c = in_c + pad_cols - out_c * stride;
					const int filter_offset = f_c + filter_cols * f_r + in_d *filter_cols*filter_rows;

					const int out_backprop_offset =
						(b * out_depth * out_rows * out_cols) +
						(in_d * out_rows * out_cols) + (out_r * out_cols) + (out_c);

					sum += (*(out_backprop + out_backprop_offset)) *
						(*(filter + filter_offset));
				}
			}
		in_backprop[thread_id] = sum;

}


void forward_depthwise_convolutional_layer_gpu(depthwise_convolutional_layer l, network net)
{
	//cuda_pull_array(l.output_gpu, l.output, l.c*l.out_h*l.out_w);//add by hjimce for debug
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);


	int size = l.out_h*l.out_w*l.batch*l.n;
	DepthwiseConv2dGPUKernelNCHW << <cuda_gridsize(size), BLOCK >> >(
		net.input_gpu,l.h,l.w,l.c,
		l.weights_gpu, l.size, l.size,
		l.stride, l.pad, l.pad,
		l.out_h, l.out_w, l.n,
		l.output_gpu, size
		);
	check_error(hipPeekAtLastError());
   /* int i;
    int k = l.size*l.size;
    int n = l.out_w*l.out_h;

	for (int b = 0; b < l.batch; ++b) {
		for (int c = 0; c<l.c; c++)
		{
			float *aoffset = l.weights_gpu + c*l.size*l.size;
			float *boffset = net.workspace;
			float *coffset = l.output_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
			float *intput_offset = net.input_gpu + c*l.h*l.w + b*l.c*l.h*l.w;
			im2col_gpu(intput_offset, 1, l.h, l.w,
				l.size, l.stride, l.pad, boffset);
			gemm_gpu(0, 0, 1, n, k, 1, aoffset, k, boffset, n, 1, coffset, n);

		}
	}*/



    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

	int m = l.n;
    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);


	//cuda_pull_array(l.output_gpu, l.output, l.c*l.out_h*l.out_w);//add by hjimce for debug

	

}


void backward_depthwise_convolutional_layer_gpu(depthwise_convolutional_layer l, network net)
{

    constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

	//cuda_pull_array(net.delta_gpu, net.delta, l.batch*l.c*l.h*l.w);
    /*int m = l.n;
    int n = l.size*l.size;
    int k = l.out_w*l.out_h;
	//pull_depthwise_convolutional_layer(l);//add by hjimce for debug

	for (int b = 0; b < l.batch; ++b) {
		for (int c = 0; c<l.c; c++)
		{



			float *aoffset = l.delta_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
			float *boffset = net.workspace;
			float *coffset = l.weight_updates_gpu + c*l.size*l.size;


			float *im = net.input_gpu + c*l.h*l.w + b*l.c*l.h*l.w;


			im2col_gpu(im, 1, l.h, l.w,
				l.size, l.stride, l.pad, boffset);
			gemm_gpu(0, 1, 1, n, k, 1, aoffset, k, boffset, k, 1, coffset, n);


			if (net.delta_gpu) {
				aoffset = l.weights_gpu + c*l.size*l.size;
				boffset = l.delta_gpu + c*l.out_h*l.out_w + b*l.n*l.out_h*l.out_w;
				coffset = net.workspace;

				gemm_gpu(1, 0, n, k, 1, 1, aoffset, n, boffset, k, 0, coffset, k);

				col2im_gpu(net.workspace, 1, l.h, l.w, l.size, l.stride, l.pad, net.delta_gpu + c*l.h*l.w + b*l.n*l.h*l.w);
			}


		}
	}*/
	
	int out_size= l.out_h*l.out_w*l.batch*l.n;
	DepthwiseConv2dBackpropFilterGPUKernelNCHW << <cuda_gridsize(out_size), BLOCK >> > (
		l.delta_gpu, l.stride, l.pad, l.pad, l.out_h, l.out_w, l.c,
		net.input_gpu, l.h, l.w, l.n,
		l.weight_updates_gpu, l.size, l.size,
		out_size);
	if (net.delta_gpu)//还在调试
	{
		int in_size = l.h*l.w*l.batch*l.n;
		DepthwiseConv2dBackpropInputGPUKernelNCHW << <cuda_gridsize(in_size), BLOCK >> > (
			l.delta_gpu, l.out_h, l.out_w, l.c,
			l.weights_gpu, l.size, l.size,
			net.delta_gpu, l.h, l.w, l.c,
			l.stride, l.pad, l.pad, in_size);

	}
	//cuda_pull_array(net.delta_gpu, net.delta, l.batch*l.c*l.h*l.w);
	//pull_depthwise_convolutional_layer(l);//add by hjimce for debug

//#endif
}
//������������
void pull_depthwise_convolutional_layer(depthwise_convolutional_layer layer)
{
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}
//������������
void push_depthwise_convolutional_layer(depthwise_convolutional_layer layer)
{
    cuda_push_array(layer.weights_gpu, layer.weights, layer.n*layer.size*layer.size);
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
}

void update_depthwise_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    int size = l.size*l.size*l.c;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, size, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }

    }else{
        axpy_gpu(size, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(size, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(size, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }


    }

}

